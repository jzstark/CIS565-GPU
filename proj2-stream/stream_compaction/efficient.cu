#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kern_scan_up(int n, int* odata, const int* idata, int d) {
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= n || index < 0) return;

            if (index % int(pow(2, d+1)) == 0) {
                //printf("(%d) index is %d: %d -- %d \n", d, index, index + int(pow(2, d + 1)) - 1, index + int(pow(2, d)) - 1);
                odata[index + int(pow(2, d + 1)) - 1] = idata[index + int(pow(2, d + 1)) - 1] + idata[index + int(pow(2, d)) - 1];
			}
        }

        __global__ void kern_scan_down(int n, int* odata, const int* idata, int d) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n || index < 0) return;

            int i, j; 
            if (index % int(pow(2,d+1)) == 0) {
                i = index + int(pow(2, d)) - 1;
                j = index + int(pow(2, d + 1)) - 1;
				//t = idata[i];
                odata[i] = idata[j];
                odata[j] = idata[i] + idata[j];
            }
        }

        __global__ void setElement(int* arr, int targetIdx, int value) {
            int tid = threadIdx.x + blockIdx.x * blockDim.x;

            if (tid == 0) {
                arr[targetIdx] = value;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO

			const int n_power = nextPow2(n);
			printf("n: %d, n_power: %d\n", n, n_power);

            int* d_odata;
            int* d_tmp_a;
            int* d_tmp_b;

            int* d_swap;
            const int D = ilog2ceil(n_power);
            hipMalloc((void**)&d_odata, n_power * sizeof(int));
            hipMalloc((void**)&d_tmp_a, n_power * sizeof(int));
            hipMalloc((void**)&d_tmp_b, n_power * sizeof(int));

            hipMemset(d_odata, 0, n_power * sizeof(int));
            hipMemset(d_tmp_a, 0, n_power * sizeof(int));
            hipMemset(d_tmp_b, 0, n_power * sizeof(int));

            // Copy data to gpu
            hipMemcpy(d_tmp_a, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_tmp_b, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // Launch kernel
            int blockSize = 256;
            int numBlocks = (n_power + blockSize - 1) / blockSize;

            for (int d = 0; d <= D - 1; d++) {
                kern_scan_up <<< numBlocks, blockSize >>> (n_power, d_tmp_b, d_tmp_a, d);
                hipMemcpy(d_tmp_a, d_tmp_b, n_power * sizeof(int), hipMemcpyDeviceToDevice);
            }

			setElement <<< 1, 1 >>> (d_tmp_a, n_power - 1, 0);

            for (int d = D - 1; d >= 0; d--) {
                kern_scan_down <<< numBlocks, blockSize >> > (n_power, d_tmp_b, d_tmp_a, d);
                hipMemcpy(d_tmp_a, d_tmp_b, n_power * sizeof(int), hipMemcpyDeviceToDevice);
                /* hipMemcpy(odata, d_tmp_b, n_power * sizeof(int), hipMemcpyDeviceToHost);
                printf("d_tmp_b:# %d", d);
                for (int i = 0; i < n_power; i++) {
                    printf("%d ", odata[i]);
                }
                printf("\n"); */
            }

            if (n_power == n) {
                hipMemcpy(odata, d_tmp_b + 1, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
                odata[n - 1] = odata[n - 2];
            } else {
                hipMemcpy(odata, d_tmp_b + 1, n * sizeof(int), hipMemcpyDeviceToHost);
            }

            // Free gpu memory
            //hipFree(d_idata);
            hipFree(d_odata);
            hipFree(d_tmp_b);
            hipFree(d_tmp_a);
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // timer().startGpuTimer();
            int* tmp1 = new int[n];
            int* tmp2 = new int[n];

            for (int i = 0; i < n; i++) {
                tmp1[i] = idata[i] == 0 ? 0 : 1;
            }
            scan(n, tmp2, tmp1);

            int idx = -1;
            for (int i = 0; i < n; i++) {
                if (tmp1[i] == 1) {
                    idx = tmp2[i] - 1;
                    odata[idx] = idata[i];
                }
            }

            delete[] tmp1;
            delete[] tmp2;
            // timer().endGpuTimer();
            return idx;
        }
    }
}
