#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;


int flag = 100;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  numObj = N; // added by me
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
  // fill dev_grdCellIndices with 0
  hipMemset(dev_gridCellStartIndices, 0, gridCellCount * sizeof(int));
  hipMemset(dev_gridCellEndIndices, 0, gridCellCount * sizeof(int));

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/

__device__ glm::vec3 rule1(int boidIndex, const glm::vec3* pos, int N) {
    glm::vec3 perceived_center(0.0f, 0.0f, 0.0f);
    int neighbor_count = 0;

    for (int i = 0; i < N; ++i) {
        if (i != boidIndex && glm::distance(pos[i], pos[boidIndex]) < rule1Distance) {
            perceived_center += pos[i];
            neighbor_count++;
        }
    }

    if (neighbor_count > 0) {
        perceived_center /= neighbor_count;
        return (perceived_center - pos[boidIndex]) * rule1Scale;
    }
    else {
        return glm::vec3(0.0f, 0.0f, 0.0f);
    }
}

__device__ glm::vec3 rule2(int boidIndex, const glm::vec3* pos, int N) {
    glm::vec3 c(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < N; ++i) {
        if (i != boidIndex && glm::distance(pos[i], pos[boidIndex]) < rule2Distance) {
            c -= (pos[i] - pos[boidIndex]);
        }
    }

    return c * rule2Scale;
}

__device__ glm::vec3 rule3(int boidIndex, const glm::vec3* pos, const glm::vec3* vel, int N) {
    glm::vec3 perceived_velocity(0.0f, 0.0f, 0.0f);
    int neighbor_count = 0;

    for (int i = 0; i < N; ++i) {
        if (i != boidIndex && glm::distance(pos[i], pos[boidIndex]) < rule3Distance) {
            perceived_velocity += vel[i];
            neighbor_count++;
        }
    }

    if (neighbor_count > 0) {
        perceived_velocity /= neighbor_count;
        return perceived_velocity * rule3Scale;
    }
    else {
        return glm::vec3(0.0f, 0.0f, 0.0f);
    }
}

__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  
    //glm::vec3 delta_vel(0.0f, 0.0f, 0.0f);
        
    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    glm::vec3 delta_vel1 = rule1(iSelf, pos, N);
    // Rule 2: boids try to stay a distance d away from each other
    glm::vec3 delta_vel2 = rule2(iSelf, pos, N);
    // Rule 3: boids try to match the speed of surrounding boids
    glm::vec3 delta_vel3 = rule3(iSelf, pos, vel, N);
    return delta_vel1 + delta_vel2 + delta_vel3;
}


__device__ glm::vec3 computeVelocityChangeNeigh(int N, int boidIndex, int* indices, const glm::vec3* pos, const glm::vec3* vel) {

	assert(N > 0);

    glm::vec3 rule1_vel(0.0f, 0.0f, 0.0f);
    glm::vec3 rule2_vel(0.0f, 0.0f, 0.0f);
    glm::vec3 rule3_vel(0.0f, 0.0f, 0.0f);
    glm::vec3 perceived_center(0.0f, 0.0f, 0.0f);
    glm::vec3 perceived_velocity(0.0f, 0.0f, 0.0f);
    glm::vec3 c(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < N; ++i) {
        int idx = indices[i];
        // if (i != boidIndex) && glm::distance(pos[idx], pos[boidIndex]) < rule1Distance) {
        perceived_center += pos[idx];
        c -= (pos[idx] - pos[boidIndex]);
        perceived_velocity += vel[i];
    }

    perceived_center /= N;
    perceived_velocity /= N;
    rule1_vel = (perceived_center - pos[boidIndex]) * rule1Scale;
    rule2_vel = c * rule2Scale;
    rule3_vel = perceived_velocity * rule3Scale;

	return rule1_vel + rule2_vel + rule3_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
    // Compute a new velocity based on pos and vel1
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
	glm::vec3 newvel = computeVelocityChange(N, index, pos, vel1);
	//TODO: clamp the speed

    // Record the new velocity into vel2. Question: why NOT vel1? - Sync 
	vel2[index] = newvel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2  
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    // Compute the grid cell index for the boid
    glm::vec3 boidPos = pos[index];
    int x = (int)((boidPos.x - gridMin.x) * inverseCellWidth);
    int y = (int)((boidPos.y - gridMin.y) * inverseCellWidth);
    int z = (int)((boidPos.z - gridMin.z) * inverseCellWidth);

    int gridIndex = gridIndex3Dto1D(x, y, z, gridResolution);

    // Store the boid index and grid cell index
    indices[index] = index;  // boid index 
	gridIndices[index] = gridIndex; // grid cell index

    return;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}


// My own method: get the neighboring cells of a boid
__device__ void getNeighboringCells(int gridResolution, glm::vec3 gridMin, float inverseCellWidth,
    float cellWidth, glm::vec3 pos, int* neighborCells, int& neighborCount, const float ruleDistance) {
    // Calculate the grid cell indices for the particle's position
    int x = (int)((pos.x - gridMin.x) * inverseCellWidth);
    int y = (int)((pos.y - gridMin.y) * inverseCellWidth);
    int z = (int)((pos.z - gridMin.z) * inverseCellWidth);

    // Calculate the range of grid cells within the specified distance
    int minX = imax(0, (int)((pos.x - gridMin.x - ruleDistance) * inverseCellWidth));
    int minY = imax(0, (int)((pos.y - gridMin.y - ruleDistance) * inverseCellWidth));
    int minZ = imax(0, (int)((pos.z - gridMin.z - ruleDistance) * inverseCellWidth));
    int maxX = imin(gridResolution - 1, (int)((pos.x - gridMin.x + ruleDistance) * inverseCellWidth));
    int maxY = imin(gridResolution - 1, (int)((pos.y - gridMin.y + ruleDistance) * inverseCellWidth));
    int maxZ = imin(gridResolution - 1, (int)((pos.z - gridMin.z + ruleDistance) * inverseCellWidth));

    // Iterate through the neighboring grid cells
    neighborCount = 0;
    for (int i = minX; i <= maxX; ++i) {
        for (int j = minY; j <= maxY; ++j) {
            for (int k = minZ; k <= maxZ; ++k) {
                neighborCells[neighborCount++] = gridIndex3Dto1D(i, j, k, gridResolution);
            }
        }
    }
    return;
}

/*
__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
    // TODO-2.1
    // Identify the start point of each cell in the gridIndices array.
    // This is basically a parallel unrolling of a loop that goes
    // "this index doesn't match the one before it, must be a new cell!"

    // TODO: Really not confident with this algorithm; perhaps need some sync 
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    if (index == 0) {
        gridCellStartIndices[particleGridIndices[0]] = 0;
    }
    else if (index == N - 1) {
		gridCellEndIndices[particleGridIndices[N - 1]] = N - 1;
    }
    else {
        int prev_num = particleGridIndices[index - 1];
        int curr_num = particleGridIndices[index];
		if (curr_num != prev_num) { // different grid
            gridCellEndIndices[prev_num] = imax(index - 1, gridCellEndIndices[prev_num]);
            gridCellStartIndices[curr_num] = imin(index, gridCellStartIndices[curr_num]);
        }
        else { // the same grid
            gridCellEndIndices[curr_num] = imax(index, gridCellEndIndices[curr_num]);
        }
    }

    return;
}

*/

/*
__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
    int* gridCellStartIndices, int* gridCellEndIndices) {
    // TODO-2.1
    // Identify the start point of each cell in the gridIndices array.
    // This is basically a parallel unrolling of a loop that goes
    // "this index doesn't match the one before it, must be a new cell!"

    // TODO: Really not confident with this algorithm; perhaps need some sync 
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= N) {
        return;
    }

    int is_start = (idx == 0   || particleGridIndices[idx] != particleGridIndices[idx - 1]);
    int is_end = (idx == N - 1 || particleGridIndices[idx] != particleGridIndices[idx + 1]);
    
	if (is_start) {
		gridCellStartIndices[particleGridIndices[idx]] = idx;
        gridCellEndIndices[particleGridIndices[idx]] = -1;
	}

	if (is_end) {
		int val = particleGridIndices[idx];
		for (int i = idx; i >= 0; i--) {
			if (particleGridIndices[i] != val) {
				gridCellEndIndices[val] = i;
				break;
			}
		}
		gridCellEndIndices[particleGridIndices[idx]] = idx;
	}

}

*/

__global__ void kernIdentifyCellStartEnd(int nBoids, int* particleGridIndices,
    int* gridCellStartIndices, int* gridCellEndIndices) {

    int i = 0;
	while (i < nBoids) {
        int val = particleGridIndices[i];
        int start = i;
		while (i + 1 < nBoids && particleGridIndices[i + 1] == val) {
			i++;
		}
        
        int end = i;
        i++;
		gridCellStartIndices[val] = start;
		gridCellEndIndices[val] = end;
	}
    return;
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
    // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
    // the number of boids that need to be checked.
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (id >= N) { return; }

    // - Identify the grid cell that this particle is in
	int gridId = gridIndex3Dto1D(
        (int)((pos[id].x - gridMin.x) * inverseCellWidth),
		(int)((pos[id].y - gridMin.y) * inverseCellWidth),
		(int)((pos[id].z - gridMin.z) * inverseCellWidth), gridResolution);
    // - Identify which cells may contain neighbors. This isn't always 8.    
	int neighborCells_rule1[27]; // possible maximum -- ??? Does not have to be; can be even larger
	int neighborCellsCount_rule1 = 0; // will be changed in getNeighboringCells()
	//TODO: Here I only use rule1Distance; should consider rule2Distance and rule3Distance
    getNeighboringCells(gridResolution, gridMin, inverseCellWidth, cellWidth, pos[id], 
        neighborCells_rule1, neighborCellsCount_rule1, rule1Distance);
    
    // - For each cell, read the start/end indices in the boid pointer array.
    int neighborNum = 0; // real size of neighbor boids
    int possibleLocalMaxBoidNum = 0; // find possible maximum size of neighbor boids
    for (int i = 0; i < neighborCellsCount_rule1; i++) {
        int cellId = neighborCells_rule1[i];
        assert(cellId >= 0 && cellId < gridResolution * gridResolution * gridResolution);

        int start = gridCellStartIndices[cellId];
        int end = gridCellEndIndices[cellId];
        if (start == INT_MAX || end == -1) {
            continue;
        }
        possibleLocalMaxBoidNum += end - start + 1;
    }
    
    int* neighborParticles = new int[possibleLocalMaxBoidNum];
    for (int i = 0; i < neighborCellsCount_rule1; i++) {
		int cellId = neighborCells_rule1[i];
		assert(cellId >= 0 && cellId < gridResolution * gridResolution * gridResolution);
        
        int start = gridCellStartIndices[cellId];
		int end = gridCellEndIndices[cellId];

        if (start == INT_MAX || end == -1) {
			continue;
		}
        assert(start >= 0 && start < N);
        assert(end >= 0 && end < N);

		for (int j = start; j <= end; j++) {
			int boidId = particleArrayIndices[j];
			if (boidId != id) {
				neighborParticles[neighborNum++] = boidId;
			}
		}
	}

    // - Access each boid in the cell and compute velocity change from
    //   the boids rules, if this boid is within the neighborhood distance.
    if (neighborNum == 0) {
		vel2[id] = vel1[id];
	} else {
		glm::vec3 newvel = computeVelocityChangeNeigh(neighborNum, id, neighborParticles, pos, vel1);
		vel2[id] = newvel;
    }
    
    // - Clamp the speed change before putting the new speed in vel2
    // TODO

    delete[] neighborParticles; // very important to release
    return;
}


__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, blockSize>>> (Boids::numObj, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (Boids::numObj, dt, dev_pos, dev_vel2);
    
    // TODO-1.2 ping-pong the velocity buffers
    glm::vec3* dev_vel_tmp;
	dev_vel_tmp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = dev_vel_tmp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
    // TODO-2.1
    // Uniform Grid Neighbor search using Thrust sort.
    // 
    // Wrap device vectors in thrust iterators for use with thrust.
  
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    // In Parallel:
    // - label each particle with its array index as well as its grid index.
    //   Use 2x width grids.
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, 
        gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    
    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices); //key
	thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices); //value
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
    
	// print the content of dev_particleArrayIndices

    if (flag == 0) {
        int* particleArrayIndices = new int[numObjects];
	    int* particleGridIndices = new int[numObjects];
	    hipMemcpy(particleArrayIndices, dev_particleArrayIndices, numObjects * sizeof(int), hipMemcpyDeviceToHost);
	    hipMemcpy(particleGridIndices,  dev_particleGridIndices,  numObjects * sizeof(int), hipMemcpyDeviceToHost);

		std::cout << numObjects << " particleArrayIndices = [";
	    for (int i = 0; i < numObjects; i++) {
		    std::cout << particleArrayIndices[i] << " ";
	    }
		std::cout << "]" << std::endl;

		std::cout << "particleGridIndices = [";
	    for (int i = 0; i < numObjects; i++) {
		    std::cout << particleGridIndices[i] << " ";
	    }
		std::cout << "]" << std::endl;
	    delete[] particleArrayIndices;
	    delete[] particleGridIndices;

        flag = 1;
	}

    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    
    kernResetIntBuffer <<<fullBlocksPerGrid, blockSize >>> (gridCellCount, dev_gridCellStartIndices, INT_MAX);
	kernResetIntBuffer <<<fullBlocksPerGrid, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
	
    //TODO: change it to sequantial algorithm for now. 
    //kernIdentifyCellStartEnd <<<fullBlocksPerGrid, blockSize >>> (gridCellCount, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    kernIdentifyCellStartEnd<<<1, 1>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

    //print gridCellEndIndices
    if (flag == 1) {
        int* gridCellEndIndices_host = new int[gridCellCount];
        hipMemcpy(gridCellEndIndices_host, dev_gridCellEndIndices, gridCellCount * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << gridCellCount << " gridCellEndIndices[" ;
        for (int i = 0; i < gridCellCount; i++) {
            std::cout << gridCellEndIndices_host[i] << " ";
        }
        std::cout << "] = " << std::endl;

        //print gridCellStartIndices
        int* gridCellStartIndices_host = new int[gridCellCount];
        hipMemcpy(gridCellStartIndices_host, dev_gridCellStartIndices, gridCellCount * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << "gridCellStartIndices[";
        for (int i = 0; i < gridCellCount; i++) {
            std::cout << gridCellStartIndices_host[i] << " ";
        }
        std::cout << "] = " << std::endl;

        delete[] gridCellEndIndices_host;
        delete[] gridCellStartIndices_host;
        
        flag = 2;
    }

    // - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered <<< fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
		dev_pos, dev_vel1, dev_vel2);

    // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);

    // - Ping-pong buffers as needed
    glm::vec3* dev_vel_tmp;
    dev_vel_tmp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = dev_vel_tmp;

    return;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
